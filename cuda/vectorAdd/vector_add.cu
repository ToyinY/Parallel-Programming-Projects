
#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 1000

__global__ void vector_add(float *out, float *a, float *b, int n) {
	for (int i = 0; i < n; i++) {
		out[i] = a[i] + b[i];
	}
}

int main(){
    float *d_a, *d_b, *d_c;
	float *h_a, *h_b, *h_c;

	h_a = (float*)malloc(N * sizeof(float));
	h_b = (float*)malloc(N * sizeof(float));
	h_c = (float*)malloc(N * sizeof(float)); 

	hipMalloc(&d_a, N * sizeof(float));
	hipMalloc(&d_b, N * sizeof(float));
	hipMalloc(&d_c, N * sizeof(float));

    // Initialize array
    for(int i = 0; i < N; i++){
        h_a[i] = 1.0f; h_b[i] = 2.0f;
    }

	// copy to host
	hipMemcpy(d_a, h_a, N * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, N * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_c, h_c, N * sizeof(float), hipMemcpyHostToDevice);

	int blocks, grids;
	blocks = 32;
	grids = (float)ceil((float)N / blocks);
	vector_add<<<grids, blocks>>>(d_c, d_a, d_b, N);
	hipMemcpy(h_c, d_c, N * sizeof(float), hipMemcpyDeviceToHost);

	int i;
	for (i = 0; i < N; i++) {
		printf("%f ", h_c[i]);
	}
	printf("\n");

	// free
	hipFree(d_a);
	hipFree(d_b);	
	hipFree(d_c);
	hipFree(h_a);
	hipFree(h_b);
	hipFree(h_c);

	return 0;
}

