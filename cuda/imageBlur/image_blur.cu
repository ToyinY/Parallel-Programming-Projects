#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>

unsigned char *input, 
unsigned char *output,
float filter;

/*******************************************************/
/*                 Cuda Error Function                 */
/*******************************************************/
inline hipError_t checkCuda(hipError_t result) {
	#if defined(DEBUG) || defined(_DEBUG)
		if (result != hipSuccess) {
			fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
			exit(-1);
		}
	#endif
		return result;
}


int imageBlur (unsigned char *input,
			   unsigned char *output,
			   unsigned int rows,
			   unsigned int cols) {

	// Allocate memory
	int size = rows * cols;
	checkCuda(hipMalloc((void**)&input_image, size * sizeof(uchar4)));
	checkCuda(hipMalloc((void**)&output_image, size * sizeof(uchar4)));
	checkCuda(hipMemset(output, 0 , size * sizeof(unsigned char)));
	
	// block and grid size
	int block_width = 16; // tile
	const dim3 block_size(block_width, block_width);	
	int blocks_x = (cols + block_width - 1) / block_width;
	int blocks_y = (rows + block_width - 1) / block_width;
	const dim3 grid_size(blocks_x, blocks_y);	

	// copy to GPU
	checkCuda(hipMemcpy(input_image, input, size * sizeof(uchar4), cusaMemcpyHostToDevice));
	hipDeviceSynchronize();

	// gaussian blur on channel
	filter_width = ;
	gaussianBlur <<<grid_size, block_size, filter_width * filter_width * sizeof(float)>>>(red, red_blurred, rows, cols, d_filer, filter_width);
	hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

	// free memory

}
