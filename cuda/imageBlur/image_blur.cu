#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>

#define TILE_SIZE 16

unsigned char *d_input;
unsigned char *d_output;
float *d_filter;

inline hipError_t checkCuda(hipError_t result) {
	#if defined(DEBUG) || defined(_DEBUG)
		if (result != cudaSuccess) {
			fprintf(stderr, "CUDA Runtime Error: %s\n", cudaGetErrorString(result));
			exit(-1);
		}
	#endif
		return result;
}

__global__ void gaussianBlur(unsigned char *input,
							 unsigned char *output,
							 unsigned int rows,
							 unsigned int cols,
							 float *filter,
							 int filter_width) {
	
	int x = blockIdx.x * TILE_SIZE + threadIdx.x;
	int y = blockIdx.y * TILE_SIZE + threadIdx.y;
	if (x >= cols || y >= rows)
		return;
	int index = y * rows + x;
	printf("thread: %d reached\n", index);
	float c = 0.0f;
	for (int fx = 0; x < filter_width; x++) {
		for (int fy = 0; fy < filter_width; y++) {
			int imagex = x + fx - filter_width / 2;
			int imagey = y + fy - filter_width / 2;
			imagex = min(max(imagex, 0), cols - 1);
			imagey = min(max(imagey, 0), rows - 1);
			c += (filter[fy * filter_width + fx] * input[imagey * cols + imagex]);
		}
	}	
	output[index] = c;
}

void imageBlur (unsigned char* h_input,
			    unsigned char* h_output,
				unsigned int rows,
			    unsigned int cols,
				float* h_filter,
				int filter_width) {

	// block and grid size
	int gridX = 1 + ((cols - 1) / TILE_SIZE);
	int gridY = 1 + ((rows - 1) / TILE_SIZE);
	dim3 dimGrid(gridX, gridY);
	dim3 dimBlock(TILE_SIZE, TILE_SIZE);

	// allocate memory and copy to GPU
	int size = rows * cols;
	checkCuda(hipMalloc((void**)&d_input, size * sizeof(unsigned char)));
	checkCuda(hipMalloc((void**)&d_output, size * sizeof(unsigned char)));
	checkCuda(hipMemset(d_output, 0, size * sizeof(unsigned char)));
	checkCuda(hipMemcpy(d_input, h_input, size * sizeof(unsigned char), hipMemcpyHostToDevice));
	checkCuda(hipMemcpy(d_filter, h_filter, filter_width * filter_width * sizeof(float), hipMemcpyHostToDevice));
	printf("rows: %d cols: %d gridx: %d gridy: %d\n", rows, cols, gridX, gridY);
	//kernel call
	gaussianBlur<<<dimGrid, dimBlock>>>(d_input, d_output, rows, cols, d_filter, filter_width);

	//copy output to host
	checkCuda(hipMemcpy(h_output, d_output, size * sizeof(unsigned char), hipMemcpyDeviceToHost));

	// free memory
	checkCuda(hipFree(d_input));
	checkCuda(hipFree(d_output));
}
